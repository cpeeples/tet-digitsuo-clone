#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "network.h"

void softmax(float *input, float *output, int size) {
    float max_val = input[0];
    for (int i = 1; i < size; i++) if (input[i] > max_val) max_val = input[i];
    float sum = 0.0f;
    for (int i = 0; i < size; i++) {
        output[i] = expf(input[i] - max_val);
        sum += output[i];
    }
    for (int i = 0; i < size; i++) output[i] /= sum;
}

__global__ void forward_hidden_kernel(const float *batch_X, const float *hidden_weights, const float *hidden_bias,
                                      float *hidden_layer, int batch_size, int input_size, int hidden_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < batch_size && j < hidden_size) {
        float sum = hidden_bias[j];
        for (int k = 0; k < input_size; k++) {
            sum += batch_X[i * input_size + k] * hidden_weights[k * hidden_size + j];
        }
        hidden_layer[i * hidden_size + j] = (sum > 0) ? sum : 0;
    }
}

__global__ void backward_dw_hidden_kernel(const float *batch_X, const float *hidden_error, float *dw_hidden,
                                          int batch_size, int input_size, int hidden_size) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    if (j < input_size && k < hidden_size) {
        float grad = 0.0f;
        for (int i = 0; i < batch_size; i++) {
            grad += batch_X[i * input_size + j] * hidden_error[i * hidden_size + k];
        }
        dw_hidden[j * hidden_size + k] = grad / batch_size;
    }
}

extern "C" void forward_pass_cuda(const Network *net, const float *batch_X, float *hidden_layer, float *output_layer) {
    float *d_batch_X, *d_hidden_weights, *d_hidden_bias, *d_hidden_layer, *d_output_weights, *d_output_bias, *d_output_layer;
    hipMalloc(&d_batch_X, BATCH_SIZE * INPUT_SIZE * sizeof(float));
    hipMalloc(&d_hidden_weights, INPUT_SIZE * HIDDEN_SIZE * sizeof(float));
    hipMalloc(&d_hidden_bias, HIDDEN_SIZE * sizeof(float));
    hipMalloc(&d_hidden_layer, BATCH_SIZE * HIDDEN_SIZE * sizeof(float));
    hipMalloc(&d_output_weights, HIDDEN_SIZE * OUTPUT_SIZE * sizeof(float));
    hipMalloc(&d_output_bias, OUTPUT_SIZE * sizeof(float));
    hipMalloc(&d_output_layer, BATCH_SIZE * OUTPUT_SIZE * sizeof(float));

    hipMemcpy(d_batch_X, batch_X, BATCH_SIZE * INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_hidden_weights, net->hidden_weights, INPUT_SIZE * HIDDEN_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_hidden_bias, net->hidden_bias, HIDDEN_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_output_weights, net->output_weights, HIDDEN_SIZE * OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_output_bias, net->output_bias, OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((BATCH_SIZE + 15) / 16, (HIDDEN_SIZE + 15) / 16);
    forward_hidden_kernel<<<blocks, threads>>>(d_batch_X, d_hidden_weights, d_hidden_bias, d_hidden_layer,
                                               BATCH_SIZE, INPUT_SIZE, HIDDEN_SIZE);
    hipDeviceSynchronize();

    hipMemcpy(hidden_layer, d_hidden_layer, BATCH_SIZE * HIDDEN_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < BATCH_SIZE; i++) {
        float tmp[OUTPUT_SIZE];
        for (int j = 0; j < OUTPUT_SIZE; j++) {
            float sum = net->output_bias[j];
            for (int k = 0; k < HIDDEN_SIZE; k++) {
                sum += hidden_layer[i * HIDDEN_SIZE + k] * net->output_weights[k * OUTPUT_SIZE + j];
            }
            tmp[j] = sum;
        }
        softmax(tmp, &output_layer[i * OUTPUT_SIZE], OUTPUT_SIZE);
    }

    hipFree(d_batch_X); hipFree(d_hidden_weights); hipFree(d_hidden_bias); hipFree(d_hidden_layer);
    hipFree(d_output_weights); hipFree(d_output_bias); hipFree(d_output_layer);
}